#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////////
// GPU-SC: GPU-accelerated version of the original Lawrence &
// Coleman shape complementarity program from CCP4.
// Luki Goldschmidt <luki@mbi.ucla.edu>, March 2011
//////////////////////////////////////////////////////////////////////
// GPU kernel functions and stubs
//////////////////////////////////////////////////////////////////////

#include "sc.h"

//////////////////////////////////////////////////////////////////////
// Collision checking GPU kernel for TrimPeripheralBand  

__global__ void _cuda_TrimPeripheralBand_kernel(
	float3 *dAccDotCoords, 
	uint nAcc, 
	float3 *dBurDotCoords, 
	char *dDotColl, 
	float r2)
{
	register int i, j, l;
	register float3 dot1;
	__shared__ char sColl[1024];
	__shared__ float3 sCoords[1024];

	sColl[threadIdx.x] = 0;
 	dot1 = dBurDotCoords[blockIdx.x*blockDim.x + threadIdx.x];

	for(i = 0; i < nAcc; i += blockDim.x) {
		__syncthreads();
		sCoords[threadIdx.x] = dAccDotCoords[i + threadIdx.x];
		__syncthreads();

		l = MIN(nAcc - i, blockDim.x);
		for(j = 0; j < l; j++) {
			register float3 dot2 = sCoords[j];
			dot2.x -= dot1.x;
			dot2.y -= dot1.y;
			dot2.z -= dot1.z;
			sColl[threadIdx.x] |= (dot2.x*dot2.x + dot2.y*dot2.y + dot2.z*dot2.z) <= r2;
		}
	}
	dDotColl[blockIdx.x*blockDim.x + threadIdx.x] = sColl[threadIdx.x];
}

//////////////////////////////////////////////////////////////////////
// Finding closest dot neighbor GPU kernel  

__global__ void _cuda_FindClosestNeighbor_kernel(
	float3 *dMyDotCoords, 
	float3 *dTheirDotCoords, 
	uint nTheirDots, 
	uint *dNeighbors)
{
	register int i, j, l;
	register float3 dot1;
	__shared__ uint sNeighbors[512];
	__shared__ float3 sCoords[512];
	float distmin = 99999.0, d2;

 	dot1 = dMyDotCoords[blockIdx.x*blockDim.x + threadIdx.x];

	for(i = 0; i < nTheirDots; i += blockDim.x) {
		__syncthreads();
		sCoords[threadIdx.x] = dTheirDotCoords[i + threadIdx.x];
		__syncthreads();

		l = MIN(nTheirDots - i, blockDim.x);
		for(j = 0; j < l; j++) {
			register float3 dot2 = sCoords[j];
			dot2.x -= dot1.x;
			dot2.y -= dot1.y;
			dot2.z -= dot1.z;
			d2 = dot2.x*dot2.x + dot2.y*dot2.y + dot2.z*dot2.z;
			if(d2 <= distmin) {
				distmin = d2;
				sNeighbors[threadIdx.x] = i+j;
			}
		}
	}
	dNeighbors[blockIdx.x*blockDim.x + threadIdx.x] = sNeighbors[threadIdx.x];
}

//////////////////////////////////////////////////////////////////////
// Stubs called from CPU code

void _cuda_TrimPeripheralBand(int x, int y, float3 *dAccDotCoords, uint nAcc, float3 *dBurDotCoords, char *dDotColl, float r2)
{
	_cuda_TrimPeripheralBand_kernel<<<x, y>>>(dAccDotCoords, nAcc, dBurDotCoords, dDotColl, r2);
}

void _cuda_FindClosestNeighbor(int x, int y, float3 *dMyDotCoords, float3 *dTheirDotCoords, uint nTheirDotsCoords, uint *dNeighbors)
{
	_cuda_FindClosestNeighbor_kernel<<<x, y>>>(dMyDotCoords, dTheirDotCoords, nTheirDotsCoords, dNeighbors);
}
